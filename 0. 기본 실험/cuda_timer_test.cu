// nvcc -o "Nomal/cuda_timer_test" "Nomal/cuda_timer_test.cu" -lpng --expt-relaxed-constexpr -lcurand -lcuda -lcudart -lcublas
// "./Nomal/cuda_timer_test"

#include <hip/hip_runtime.h>
#include <iostream>
#include <functional> // std::function

void cuda_timmer(const std::string& msg, std::function<void()> f) {
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    f();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << msg << " Elapsed time: " << elapsedTime << " ms" << std::endl;
}

int main() {

    // Nomal

    hipEvent_t start0, stop0;
    float elapsedTime;

    hipEventCreate(&start0);
    hipEventCreate(&stop0);
    hipEventRecord(start0, 0);

    // 실행할 코드

    hipEventRecord(stop0, 0);
    hipEventSynchronize(stop0);
    hipEventElapsedTime(&elapsedTime, start0, stop0);

    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;


    // Lamda function
    
    cuda_timmer("Execution time", []() {
        // 실행할 코드
    });

    return 0;

}
